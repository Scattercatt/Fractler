
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>



__global__ void vectorAdd(int* c, const int* a, const int* b)
{

}


int main(int argc, char* argv[])
{
    
    /*const int EXPECTED_ARG_COUNT = 13;

    if (argc < EXPECTED_ARG_COUNT)
    {
        printf("Must specify all args for render_column.c");
        return 1;
    }*/

    // Create pointers
    hipDoubleComplex* point_one = (hipDoubleComplex*)malloc(2 * sizeof(hipDoubleComplex));
    hipDoubleComplex* point_two = (hipDoubleComplex*)malloc(2 * sizeof(hipDoubleComplex));
    
    point_one[0] = make_hipDoubleComplex(strtod(argv[0], NULL), strtod(argv[1], NULL));
    point_two[0] = make_hipDoubleComplex(strtod(argv[2], NULL), strtod(argv[3], NULL));

    // Arg 5 is the size of the desired output array
    int img_size = atoi(argv[5]);

    // Arg 6 is the column number
    int column_to_render = atoi(argv[6]);

    if (column_to_render >= img_size)
    {
       printf("Render column cannot be greater than or equal to image size!");
       return 1;
    }

    // Arg 7 is the max iteration count
    int max_iterations = atoi(argv[7]);

    printf("%d",max_iterations);

    //// Arg 8 is the fractal to render. 
    //int fractal_int = atoi(argv[8]);
    //FractalFuncPtr selected_fractal = get_fractal_from_int(fractal_int);

    //// Arg 9 is the bailout condition
    //int bailout_int = atoi(argv[9]);
    //BailoutFuncPtr selected_bailout = get_bailout_from_int(bailout_int);

    //// Arg 10 is whether or not the image is a julia
    //bool is_julia = (atoi(argv[10]) != 0);

    //// Arg 11 and 12 are the julia points.
    //int julia_point_real, julia_point_imag;
    //if (is_julia)
    //{
    //    julia_point_real = atoi(argv[11]);
    //    julia_point_imag = atoi(argv[12]);
    //}


    //// Column separation amount
    //double column_separation_amount = (creal(p2) - creal(p1)) / (float)img_size;
    //double row_separation_amount = (cimag(p2) - cimag(p1)) / (float)img_size;

    //// Column double to render across
    //double column_to_render_down = column_separation_amount * column_to_render + creal(p1);


    //// Define data return var
    //unsigned short calculated_iterations[img_size];

    //// Begin fractal calculation
    //int pixel;
    //double row;
    //for (row = cimag(p1), pixel = 0; row < cimag(p2); row += row_separation_amount, pixel++)
    //{


    //    // Set initial Z and C based on is_julia
    //    cdouble Z, C;
    //    if (!is_julia)
    //    {
    //        Z = C_ZERO;

    //        C = column_to_render_down + row * I;
    //    }
    //    else
    //    {
    //        Z = column_to_render_down + row * I;

    //        C = julia_point_real + julia_point_imag;
    //    }

    //    int its = 0;
    //    while (its < max_iterations)
    //    {
    //        bool bailout_succeeded = selected_bailout(Z, C);
    //        if (bailout_succeeded)
    //        {
    //            break;
    //        }
    //        Z = selected_fractal(Z, C);

    //        its++;
    //    }

    //    calculated_iterations[pixel] = its;
    //}

    //for (int i = 0; i < img_size; i++)
    //{
    //    printf("%d ", calculated_iterations[i]);
    //}

    return 0;
}