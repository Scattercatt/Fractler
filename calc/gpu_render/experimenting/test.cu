
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>

__global__ void do_stuff(int *a, int N)
{
    short i = threadIdx.x;
    if (i < N)
    {
        a[i] = a[i] * 1000;
    }
}
int main()
{
    int N = 100;
    int* h_a = (int*) malloc(N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i * i;
    }

    int* d_a;

    printf("Malloc memory to device\n");
    hipMalloc((void**)&d_a, N * sizeof(int));


    printf("Copy memory to device\n");
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid_size(1); 
    dim3 block_size(N);

    printf("Run kernal\n");
    do_stuff<<<grid_size, block_size>>>(d_a, N);

    printf("Copy memory back\n");
    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);


    printf("Print results in the form of h_a\n");
    for (int i = 0; i < N; i++)
    {
        printf("%d: ", h_a[i]);
    }

    hipFree(d_a);
    free(h_a);
}